﻿#include "file_system.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#define valid_bit(i) (i) * fs->FCB_SIZE + 28 + fs->SUPERBLOCK_SIZE
#define size_bit(i)  (i) * fs->FCB_SIZE + 26 + fs->SUPERBLOCK_SIZE
#define addr_bit(i)  (i) * fs->FCB_SIZE + 24 + fs->SUPERBLOCK_SIZE
#define mtime_bit(i) (i) * fs->FCB_SIZE + 22 + fs->SUPERBLOCK_SIZE
#define ctime_bit(i) (i) * fs->FCB_SIZE + 20 + fs->SUPERBLOCK_SIZE
#define name_bit(i)  (i) * fs->FCB_SIZE + 0  + fs->SUPERBLOCK_SIZE

__device__ __managed__ u32 gtime = 0;
__device__ __managed__ u32 file_number = 0;
__device__ __managed__ u32 gsize = 0; // last block number

__device__ void FCB_init(FileSystem *fs)
{
  // 0-19 name
  // 20-21 create time
  // 22-23 modified time
  // 24-25 address
  // 26-27 size
  // 28 valid bit
  for (int i=0;i<fs->FCB_ENTRIES;i++)//set valid bit
  {
    fs->volume[valid_bit(i)] = 0xff;
  }
}
__device__ void SUPERBLOCK_init(FileSystem *fs)
{
  for (int i=0;i<fs->SUPERBLOCK_SIZE;i++)
  {
    fs->volume[i]=0;
  }
}

__device__ void fs_init(FileSystem *fs, uchar *volume, int SUPERBLOCK_SIZE,
							int FCB_SIZE, int FCB_ENTRIES, int VOLUME_SIZE,
							int STORAGE_BLOCK_SIZE, int MAX_FILENAME_SIZE, 
							int MAX_FILE_NUM, int MAX_FILE_SIZE, int FILE_BASE_ADDRESS)
{
  // init variables
  fs->volume = volume;

  // init constants
  fs->SUPERBLOCK_SIZE = SUPERBLOCK_SIZE;
  fs->FCB_SIZE = FCB_SIZE;
  fs->FCB_ENTRIES = FCB_ENTRIES;
  fs->STORAGE_SIZE = VOLUME_SIZE;
  fs->STORAGE_BLOCK_SIZE = STORAGE_BLOCK_SIZE;
  fs->MAX_FILENAME_SIZE = MAX_FILENAME_SIZE;
  fs->MAX_FILE_NUM = MAX_FILE_NUM;
  fs->MAX_FILE_SIZE = MAX_FILE_SIZE;
  fs->FILE_BASE_ADDRESS = FILE_BASE_ADDRESS;

  SUPERBLOCK_init(fs);
  FCB_init(fs);
}

__device__ void string_print(char *s)
{
  while(*s != '\0'){
    printf("%c",*s);
    s++;
  }
  // printf("\n");
}

__device__ bool check_name(char *s1, char *s2)
{
  while(*s1 != '\0' && *s2 != '\0' && *s1 == *s2)
  {
    s1++;
    s2++;
  }

  if((*s1 == '\0')&&(*s2 == '\0'))
  {
    return true;
  }
  return false;
}


__device__ bool update_file_name(FileSystem *fs, char *s, int file)
{
  int count=0;
  while(*s != '\0')
  {
    fs->volume[name_bit(file)+count] = *s;
    s++;
    count++;
    if(count == fs->MAX_FILENAME_SIZE)
    {
      printf("ERROR: file name too large.\n");
      return true;
    }
  }
  fs->volume[name_bit(file)+count] = '\0';
  return false;
}
__device__ void modified_FCB(FileSystem *fs, int file, int size)
{
  //modified time
  fs->volume[mtime_bit(file)+0] = (gtime>>8) & 0xff;
  fs->volume[mtime_bit(file)+1] = (gtime>>0) & 0xff;

  //modified size
  fs->volume[size_bit(file)+0] = (size>>8) & 0xff;
  fs->volume[size_bit(file)+1] = (size>>0) & 0xff;

  //valid bit set to 0
  fs->volume[valid_bit(file)] = 0;
}


__device__ u32 fs_open(FileSystem *fs, char *s, int op)
{
	int empty = -1;
  for(int i = 0; i < fs->FCB_ENTRIES; i++)// find file
  {
    if(fs->volume[valid_bit(i)] != 0xff)// valid? Yes
    {
      if(check_name((char *)&fs->volume[name_bit(i)], s))// The file? Yes
      {
        return i;
      }
    }
    else // valid? No
    {
      if(empty == -1)// First empty address
      {
        empty = i;
      }
    }
  }

//  if(op == G_READ)printf("ERROR: No such file.\n");

  if(empty == -1)// cant build another file
  {
    printf("ERROR: files number reach the maximun.\n");
    return 0xffffffff;
  }
  else//create new file to write
  {
    if(update_file_name(fs, s, empty))// cant write file name
    {
      return 0xffffffff;
    }
    modified_FCB(fs, empty, 0);

    fs->volume[ctime_bit(empty)+0] = (gtime>>8) & 0xff;
    fs->volume[ctime_bit(empty)+1] = (gtime>>0) & 0xff;

    gtime++;
    file_number++;
  }
  return empty;
}


__device__ void fs_read(FileSystem *fs, uchar *output, u32 size, u32 fp)
{
	if( fp >= 1024 && fp < 0 && fs->volume[valid_bit(fp)] == 0xff)
  {
    printf("ERROR: error fp.\n");
    return;
  }

  int file_size = (fs->volume[size_bit(fp)+0]<<8) + fs->volume[size_bit(fp)+1];
  int file_addr = (fs->volume[addr_bit(fp)+0]<<8) + fs->volume[addr_bit(fp)+1];
  file_addr *= fs->STORAGE_BLOCK_SIZE;

  if(file_size < size)
  {
    // uchar *s = &fs->volume[file_addr + fs->FILE_BASE_ADDRESS];
    // printf("fp=%d size = %d require size = %d   ",(int)fp, file_size, size);
    // string_print((char *)s);

    printf("ERROR: require size is too large\n");
    return;
  }

  for(int i=0; i < size; i++)
  {
    output[i] = fs->volume[file_addr + i + fs->FILE_BASE_ADDRESS]; 
  }
}

__device__ void set_superblock_bit(FileSystem *fs, u32 addr, int bit)
{
  int i = addr >> 3;
  int j = addr & 0x07;
  u_int update = (1<<j);
  if(bit==1)
  {
    fs->volume[i] |= update;
  }
  else
  {
    fs->volume[i] &= ~(update);
  }
}

__device__ void remove_file(FileSystem *fs, u32 fp)
{
  int size = (fs->volume[size_bit(fp)+0]<<8) + fs->volume[size_bit(fp)+1];
  int addr = (fs->volume[addr_bit(fp)+0]<<8) + fs->volume[addr_bit(fp)+1];
  addr *= fs->STORAGE_BLOCK_SIZE;

  for (int i = 0; i < size; i++)//release fp addr
  {
    fs->volume[i + addr + fs->FILE_BASE_ADDRESS] = 0;
  }
  int block_number = (size+fs->STORAGE_BLOCK_SIZE-1) / fs->STORAGE_BLOCK_SIZE;
  addr /= fs->STORAGE_BLOCK_SIZE;
  for (int i = 0; i < block_number; i++)// set superblock of fp to 0
  {
    set_superblock_bit(fs, addr+i, 0);
  }

  for (int i = addr + block_number; i < gsize; i++)// move addr after fp block_number front
  {
    for (int j = 0; j < fs->STORAGE_BLOCK_SIZE; j++)// size
    {
      fs->volume[(i-block_number)*fs->STORAGE_BLOCK_SIZE+j+fs->FILE_BASE_ADDRESS] = fs->volume[i*fs->STORAGE_BLOCK_SIZE+j+fs->FILE_BASE_ADDRESS];
    }
    set_superblock_bit(fs, i-block_number, 1);
    set_superblock_bit(fs, i, 0);
  }

  for (int i = 0; i < fs->FCB_ENTRIES; i++)//search file
  {
    if(fs->volume[valid_bit(i)]!=0xff)// the file vaild
    {
      int i_addr = (fs->volume[addr_bit(i)+0]<<8) + fs->volume[addr_bit(i)+1];
      if(i_addr > addr)// reset addr pointer of other file (after fp)
      {
        i_addr -= block_number;
        fs->volume[addr_bit(i)+0] = (i_addr>>8) & 0xff;
        fs->volume[addr_bit(i)+1] = (i_addr>>0) & 0xff;
      }
    }
  }
  gsize -= block_number;// total number 
}

__device__ u32 fs_write(FileSystem *fs, uchar* input, u32 size, u32 fp)
{
	if( fp >= 1024 && fp < 0 && fs->volume[valid_bit(fp)] == 0xff)
  {
    printf("ERROR: error fp.\n");
    return 0xffffffff;
  }
  if(size > fs->MAX_FILE_SIZE)
  {
    printf("ERROR: file size too large.\n");
    return 0xffffffff;
  }
  
  int old_size = (fs->volume[size_bit(fp)+0]<<8) + fs->volume[size_bit(fp)+1];
  int addr;
  
  if (old_size == 0)// set in end
  {
    addr = gsize;
  }
  else if((old_size + fs->STORAGE_BLOCK_SIZE-1)%32 != (size+ fs->STORAGE_BLOCK_SIZE-1)%32)//storage space not change
  {
    addr = (fs->volume[addr_bit(fp)+0]<<8) + fs->volume[addr_bit(fp)+1];
  }
  else// cant fit in well
  {
    remove_file(fs, fp);
    addr = gsize;
  }
  /*reset modified time*/
  gtime++;
  fs->volume[mtime_bit(fp)+0] = (gtime>>8) & 0xff;
  fs->volume[mtime_bit(fp)+1] = (gtime>>0) & 0xff;
  
  /*reset address*/
  fs->volume[addr_bit(fp)+0] = (addr>>8) & 0xff;
  fs->volume[addr_bit(fp)+1] = (addr>>0) & 0xff;
  
  /*reset size*/
  fs->volume[size_bit(fp)+0] = (size>>8) & 0xff;
  fs->volume[size_bit(fp)+1] = (size>>0) & 0xff;
  int block_number = (size + fs->STORAGE_BLOCK_SIZE-1)/fs->STORAGE_BLOCK_SIZE;
  for (int i = 0; i < block_number; i++)//superblock valid
  {
    set_superblock_bit(fs, addr+i, 1);
  }
  if(gsize == addr) gsize += block_number;

  addr *= fs->STORAGE_BLOCK_SIZE;
  for (int i = 0; i < size ; i++)//storage input
  {
    fs->volume[addr + i + fs->FILE_BASE_ADDRESS] = input[i];
  }

  // printf("fp=%d size=%d %d\n",(int)fp,size,(fs->volume[size_bit(fp)+0]<<8) + fs->volume[size_bit(fp)+1]);
  // printf("gsize=%d\n",gsize);
}

__device__ bool cmp(FileSystem *fs, int a, int b, int op)
{
  if (a == -1) return true;
  if (op == LS_D)
  {
    int a_time = (fs->volume[mtime_bit(a)+0]<<8) + (fs->volume[mtime_bit(a)+1]);
    int b_time = (fs->volume[mtime_bit(b)+0]<<8) + (fs->volume[mtime_bit(b)+1]);
    if(a_time != b_time) return (a_time < b_time);
  }
  else 
  {
    int a_size = (fs->volume[size_bit(a)+0]<<8) + (fs->volume[size_bit(a)+1]);
    int b_size = (fs->volume[size_bit(b)+0]<<8) + (fs->volume[size_bit(b)+1]);
    if (a_size != b_size) return (a_size < b_size);
  }

  int a_ctime = (fs->volume[ctime_bit(a)+0]<<8) + (fs->volume[ctime_bit(a)+1]);
  int b_ctime = (fs->volume[ctime_bit(b)+0]<<8) + (fs->volume[ctime_bit(b)+1]);
  return a_ctime > b_ctime;
}

__device__ void fs_gsys(FileSystem *fs, int op)
{
  if (op == LS_D)
  {
    printf("===sort by modified time===\n");
  }
  if (op == LS_S)
  {
    printf("===sort by file size===\n");
  }
  for (int cnt = 0; cnt < file_number; cnt++)
  {
    int first=-1;
    for (int i = 0; i < fs->FCB_ENTRIES; i++)
    {
      if (fs->volume[valid_bit(i)] == 0)
      {
        if (cmp(fs, first, i, op))
        {
          first=i;
        }
      }
    }
    if (first != -1)
    {
      string_print((char*)&fs->volume[name_bit(first)]);
      if (op == LS_S) printf(" %d",(fs->volume[size_bit(first)]<<8)+(fs->volume[size_bit(first)+1]));
      printf("\n");
      fs->volume[valid_bit(first)]=0x01;
    }
  }

  for (int i = 0; i < fs->FCB_ENTRIES; i++)
  {
    if(fs->volume[valid_bit(i)]==0x01)
    {
      fs->volume[valid_bit(i)]=0x00;
    }
  }
  return;
}

__device__ void fs_gsys(FileSystem *fs, int op, char *s)
{
  int fp = -1;
  for(int i = 0; i < fs->FCB_ENTRIES; i++)// find file
  {
    if(fs->volume[valid_bit(i)] != 0xff)// valid? Yes
    {
      if(check_name((char *)&fs->volume[name_bit(i)], s))// The file? Yes
      {
        fp = i;
        break;
      }
    }
  }
  if (fp == -1)
  {
    printf("ERROR: haven't find the file.");
    return;
  }
  remove_file(fs, (u32)fp);
  fs->volume[valid_bit(fp)]=0xff;
}
